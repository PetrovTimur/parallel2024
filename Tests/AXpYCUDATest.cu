#include "hip/hip_runtime.h"
#include "thrust/device_vector.h"
#include "Solver/Kernels/mathfunc.cuh"
#include <iostream>
#include <vector>


void axpy_gpu(float a, float *x, float *y, int size, float *res) {
    int threadsPerBlock = 256;
    int blocksPerGrid = (size + threadsPerBlock - 1) / threadsPerBlock;

    float *d_x, *d_y, *d_res;
    hipMalloc((void**)&d_x, size * sizeof(float));
    hipMalloc((void**)&d_y, size * sizeof(float));
    hipMalloc((void**)&d_res, size * sizeof(float));

    hipMemcpy(d_x, x, size * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_y, y, size * sizeof(float), hipMemcpyHostToDevice);

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    hipEventRecord(start);
    axpy<<<blocksPerGrid, threadsPerBlock>>>(a, d_x, d_y, size, d_res);
    hipEventRecord(stop);

    hipEventSynchronize(stop);
    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);
    std::cout << 2 * size / 1e9 / (milliseconds / 1000.0) << ", " << milliseconds / 1000.0 << std::endl;

    hipEventDestroy(start);
    hipEventDestroy(stop);

    hipMemcpy(res, d_res, size * sizeof(float), hipMemcpyDeviceToHost);

    hipFree(d_x);
    hipFree(d_y);
    hipFree(d_res);
}

int main() {
    int size = 100000000;
    float a = 2.0;
    std::vector<float> x(size, 1.0);
    std::vector<float> y(size, 2.0);
    std::vector<float> res(size);

    axpy_gpu(a, x.data(), y.data(), size, res.data());

    // Verify the result & print first error if any
    bool correct = true;
    for (int i = 0; i < size; ++i) {
        float expected = a * x[i] + y[i];
        if (res[i] != expected) {
            std::cerr << "Mismatch at index " << i << ": " << res[i] << " != " << expected << std::endl;
            correct = false;
            break;
        }
    }

    if (correct) {
        std::cout << "AXPY test passed!" << std::endl;
    } else {
        std::cout << "AXPY test failed!" << std::endl;
    }

    return 0;
}