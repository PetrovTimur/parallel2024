#include "hip/hip_runtime.h"
#include "thrust/device_vector.h"
#include "Solver/Kernels/mathfunc.cuh"
#include <iostream>
#include <vector>


void axpy_gpu(float a, float *x, float *y, int size, float *res) {
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    hipEventRecord(start);
    axpy<<<204, 256>>>(a, x, y, size, res);
    hipEventRecord(stop);

    hipEventSynchronize(stop);
    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);
    std::cout << 2 * size / 1e9 / (milliseconds / 1000.0) << ", " << milliseconds / 1000.0 << std::endl;

    hipEventDestroy(start);
    hipEventDestroy(stop);
}

int main(int argc, char *argv[]) {
    int size = std::atoi(argv[1]);
    float a = 2.0;
    std::vector<float> x(size, 1.0);
    std::vector<float> y(size, 2.0);
    std::vector<float> res(size);

    thrust::device_vector<float> d_x = x;
    thrust::device_vector<float> d_y = y;
    thrust::device_vector<float> d_res(size);

    axpy_gpu(a, d_x.data().get(), d_y.data().get(), size, d_res.data().get());

    hipMemcpy(res.data(), d_res.data().get(), size*sizeof(float), hipMemcpyDeviceToHost);


    // Verify the result & print first error if any
    bool correct = true;
    for (int i = 0; i < size; ++i) {
        float expected = a * x[i] + y[i];
        if (res[i] != expected) {
            std::cerr << "Mismatch at index " << i << ": " << res[i] << " != " << expected << std::endl;
            correct = false;
            break;
        }
    }

    if (correct) {
        std::cout << "AXPY test passed!" << std::endl;
    } else {
        std::cout << "AXPY test failed!" << std::endl;
    }

    return 0;
}