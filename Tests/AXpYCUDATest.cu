#include "hip/hip_runtime.h"
#include "thrust/device_vector.h"
#include "Solver/Kernels/mathfunc.cuh"
#include <iostream>
#include <vector>


void axpy_gpu(double a, double *x, double *y, int size, double *res) {
    int threadsPerBlock = 256;
    int blocksPerGrid = (size + threadsPerBlock - 1) / threadsPerBlock;

    double *d_x, *d_y, *d_res;
    hipMalloc((void**)&d_x, size * sizeof(double));
    hipMalloc((void**)&d_y, size * sizeof(double));
    hipMalloc((void**)&d_res, size * sizeof(double));

    hipMemcpy(d_x, x, size * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(d_y, y, size * sizeof(double), hipMemcpyHostToDevice);

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    hipEventRecord(start);
    axpy<<<blocksPerGrid, threadsPerBlock>>>(a, d_x, d_y, size, d_res);
    hipEventRecord(stop);

    hipEventSynchronize(stop);
    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);
    std::cout << 2 * size / 1e9 / (milliseconds / 1000.0) << ", " << milliseconds / 1000.0 << std::endl;

    hipEventDestroy(start);
    hipEventDestroy(stop);

    hipMemcpy(res, d_res, size * sizeof(double), hipMemcpyDeviceToHost);

    hipFree(d_x);
    hipFree(d_y);
    hipFree(d_res);
}

int main() {
    int size = 100000000;
    double a = 2.0;
    std::vector<double> x(size, 1.0);
    std::vector<double> y(size, 2.0);
    std::vector<double> res(size);

    axpy_gpu(a, x.data(), y.data(), size, res.data());

    // Verify the result & print first error if any
    bool correct = true;
    for (int i = 0; i < size; ++i) {
        double expected = a * x[i] + y[i];
        if (res[i] != expected) {
            std::cerr << "Mismatch at index " << i << ": " << res[i] << " != " << expected << std::endl;
            correct = false;
            break;
        }
    }

    if (correct) {
        std::cout << "AXPY test passed!" << std::endl;
    } else {
        std::cout << "AXPY test failed!" << std::endl;
    }

    return 0;
}