#include "thrust/device_vector.h"
#include <thrust/host_vector.h>
#include "Solver/Kernels/mathfunc.cuh"
#include "hip/hip_runtime.h"
#include <iostream>
#include <omp.h>
#include <vector>

int main() {
    int blocks = 128;
    int threads = 256;
    int size = 100000000;
    float a = 1.0f;

    thrust::host_vector<float> x(size, a);
    thrust::device_vector<float> d_x(size);
    thrust::device_vector<float> d_y(blocks);
    d_x = x;

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    hipEventRecord(start);
    reduce0<<<blocks, threads, threads*sizeof(float)>>>(d_x.data().get(), d_y.data().get(), size);
    reduce0<<<1, blocks, blocks*sizeof(float)>>>(d_y.data().get(), d_x.data().get(), blocks);
    hipEventRecord(stop);

    hipEventSynchronize(stop);
    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);
    std::cout << 2 * size / 1e9 / (milliseconds / 1000.0) << " GFLOPS, " << milliseconds << " ms" << std::endl;

    float res = d_x[0];
    d_x[0] = a;

    // Verify the result & print first error if any
    bool correct = (res == size * a);

    std::cout << res << std::endl;
    std::cout << size * a << std::endl;


    if (correct) {
        std::cout << "Reduce test passed!" << std::endl;
    } else {
        std::cout << "Reduce test failed!" << std::endl;
    }

    std::cout << std::endl << "-----------------" << std::endl << std::endl;


    blocks = (size + threads - 1) / threads;
    thrust::device_vector<float> d_intermediate(blocks);
    d_y.resize(blocks);

    hipEventRecord(start);
    reduce1<<<blocks, threads>>>(d_x.data().get(), d_y.data().get(), size);

    // std::cout << d_y[0] << d_y[1] << std::endl;

    int s = blocks;
    while (s > 1) {
        blocks = (s + threads - 1) / threads;

        hipMemcpy(d_intermediate.data().get(), d_y.data().get(), s * sizeof(float), hipMemcpyDeviceToDevice);
        reduce1<<<blocks, threads>>>(d_intermediate.data().get(), d_y.data().get(), s);

        s = (s + threads - 1) / threads;
    }
    hipEventRecord(stop);

    hipEventSynchronize(stop);
    milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);
    std::cout << 2 * size / 1e9 / (milliseconds / 1000.0) << " GFLOPS, " << milliseconds << " ms" << std::endl;

    res = d_y[0];

    // Verify the result & print first error if any
    correct = (res == size * a);

    std::cout << res << std::endl;
    std::cout << size * a << std::endl;

    if (correct) {
        std::cout << "Reduce test passed!" << std::endl;
    } else {
        std::cout << "Reduce test failed!" << std::endl;
    }

    // hipDeviceProp_t prop{};
    // int device;
    // hipGetDevice(&device);
    // hipGetDeviceProperties(&prop, device);
    //
    // std::cout << "Device " << prop.name << std::endl;
    // std::cout << prop.major << "." << prop.minor << std::endl;
    // std::cout << prop.maxThreadsPerBlock << std::endl;
    // std::cout << prop.maxGridSize[0] <<std::endl;
    // std::cout << prop.maxBlocksPerMultiProcessor << std::endl;
    // std::cout << prop.maxThreadsPerMultiProcessor << std::endl;


    return 0;
}