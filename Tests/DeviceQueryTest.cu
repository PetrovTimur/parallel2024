
#include <hip/hip_runtime.h>
#include <iostream>

int main() {
    hipDeviceProp_t prop{};
    int device;
    hipGetDevice(&device);
    hipGetDeviceProperties(&prop, device);

    std::cout << "Device " << prop.name << std::endl;
    std::cout << prop.major << "." << prop.minor << std::endl;
    std::cout << prop.maxThreadsPerBlock << std::endl;
    std::cout << prop.maxBlocksPerMultiProcessor << std::endl;
    std::cout << prop.multiProcessorCount <<std::endl;
    std::cout << prop.maxThreadsPerMultiProcessor << std::endl;

    std::cout << std::endl << "-----------------" << std::endl << std::endl;

}
