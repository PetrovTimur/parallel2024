
#include <hip/hip_runtime.h>
#include <iostream>

int main() {
    hipDeviceProp_t prop{};
    int device;
    hipGetDevice(&device);
    hipGetDeviceProperties(&prop, device);

    std::cout << "Device: " << prop.name << ", Compute Capability: " << prop.major << "." << prop.minor << std::endl;
    std::cout << "Max threads per block: " << prop.maxThreadsPerBlock << std::endl;
    std::cout << "Max blocks per SM: " << prop.maxBlocksPerMultiProcessor << std::endl;
    std::cout << "SM count: " << prop.multiProcessorCount << std::endl;
    std::cout << "Max threads per SM: " << prop.maxThreadsPerMultiProcessor << std::endl;

}
