#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "thrust/device_vector.h"

__device__ inline int ceilPow2(const unsigned int n) {
    // early out if already power of two
    if (0 == (n & (n - 1))) {
        return n;
    }

    int exp;
    frexp(static_cast<float>(n), &exp);
    return (1 << exp);
}

__global__ void axpy(const float a, const float *x, const float *y, const int size, float *res) {
    const unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;
    const unsigned int stride = blockDim.x * gridDim.x;

    for (unsigned int k = i; k < size; k += stride) {
        res[k] = a * x[k] + y[k];
    }
}

__global__ void reduce0(const float *x, float *y, const int N) {
    extern __shared__ float tsum[];

    const unsigned int tid = threadIdx.x;
    const unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;
    const unsigned int stride = gridDim.x * blockDim.x;

    tsum[tid] = 0.0f;

    for (unsigned int k = i; k < N; k += stride) {
        tsum[tid] += x[k];
    }

    __syncthreads();

    int block2 = ceilPow2(blockDim.x);

    for (unsigned int k = block2 / 2; k > 0; k >>= 1) {
        if (tid < k && tid + k < blockDim.x) {
            tsum[tid] += tsum[tid + k];
        }
        __syncthreads();
    }

    if (tid == 0) {
        y[blockIdx.x] = tsum[0];
    }
}

__global__ void reduce1(const float *g_idata, float *g_odata, const unsigned int n) {
    extern __shared__ float sdata[];

    // load shared mem
    const unsigned int tid = threadIdx.x;
    const unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;

    sdata[tid] = (i < n) ? g_idata[i] : 0;

    __syncthreads();


    // do reduction in shared mem
    for (unsigned int s = blockDim.x / 2; s > 0; s >>= 1) {
        if (tid < s) {
            sdata[tid] += sdata[tid + s];
        }

        __syncthreads();
    }

    // write result for this block to global mem
    if (tid == 0) g_odata[blockIdx.x] = sdata[0];
}

__global__ void spMV(const int *ia, const int *ja, const float *a, const float *x, float *y, const int size) {
    const unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;
    const unsigned int stride = gridDim.x * blockDim.x;

    for (unsigned int k = i; k < size; k += stride) {
        float sum = 0.0;
        for (unsigned int p = ia[k]; p < ia[k + 1]; p++) {
            const unsigned int j = ja[p];
            const float a_ij = a[p];
            sum += x[j] * a_ij;
        }

        y[k] = sum;
    }
}

__global__ void dot(const float *x, const float *y, float *z, const int N) {
    extern __shared__ float tsum[];

    const unsigned int tid = threadIdx.x;
    const unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;
    const unsigned int stride = gridDim.x * blockDim.x;

    tsum[tid] = 0.0f;

    for (unsigned int k = i; k < N; k += stride) {
        tsum[tid] += x[k] * y[k];
    }

    __syncthreads();

    for (unsigned int k = blockDim.x / 2; k > 0; k /= 2) {
        if (tid < k) {
            tsum[tid] += tsum[tid + k];
        }
        __syncthreads();
    }

    if (tid == 0) {
        z[blockIdx.x] = tsum[0];
    }
}

__global__ void multiply(const float *x, const float *y, float *z, const int N) {
    const unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;
    const unsigned int stride = gridDim.x * blockDim.x;

    for (unsigned int k = i; k < N; k += stride) {
        z[k] = x[k] * y[k];
    }
}

