#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "thrust/device_vector.h"
#include "Solver/Kernels/mathfunc.cuh"
#include <Utilities/logger.h>
#include "solvers.cuh"
#include "omp.h"


#define checkCudaErrors(val) check((val), #val, __FILE__, __LINE__)
void check(hipError_t err, const char* const func, const char* const file,
           const int line)
{
    if (err != hipSuccess)
    {
        std::cerr << "CUDA Runtime Error at: " << file << ":" << line
                  << std::endl;
        std::cerr << hipGetErrorString(err) << " " << func << std::endl;
        // We don't exit when we encounter CUDA errors in this example.
        std::exit(EXIT_FAILURE);
    }
}

#define getLastCudaError(msg) checkLast(msg, __FILE__, __LINE__)

inline void checkLast(const char *errorMessage, const char *file,
                               const int line) {
    hipError_t err = hipGetLastError();

    if (hipSuccess != err) {
        fprintf(stderr,
                "%s(%i) : getLastCudaError() CUDA error :"
                " %s : (%d) %s.\n",
                file, line, errorMessage, static_cast<int>(err),
                hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
}

int solve(const int *ia, const int *ja, const float *a, const float *b, const float *diag, int size, float *res, const double eps, const int maxit) {
    const int N = size - 1;
    const int blocks = 128;
    const int threads = 256;

    thrust::device_vector<float> z(N);
    thrust::device_vector<float> p(N);
    thrust::device_vector<float> q(N);
    thrust::device_vector<float> x(N);
    thrust::device_vector<float> r(b, b + N);
    thrust::device_vector<float> diag_gpu(diag, diag + N);

    thrust::device_vector<float> rho(2);
    thrust::device_vector<int> ia_gpu(ia, ia + N + 1);
    thrust::device_vector<int> ja_gpu(ja, ja + ia[N]);
    thrust::device_vector<float> a_gpu(a, a + ia[N]);

    thrust::device_vector<float> vec_buf(blocks);

    float *buf_gpu, *norm_gpu;
    hipMalloc(&buf_gpu, sizeof(float));
    hipMalloc(&norm_gpu, sizeof(float));
    int k = 0;
    float buf, norm;

    // while (rho[1] > eps * eps && k < maxit) {
    //     if (k > 1) {
    //         beta = rho[1] / rho[0];
    //     } else {
    //         p = z;cuda-memcheck ./bin/CGSolver --Nx=4 --Ny=4 --K1=2 --K2=3
    //     }
    //
    //
    //
    //     k++;
    // }

    std::cout << "Starting: ";

    do {
        k++;

        LOG_INFO << "Iteration " << k << std::endl;
        std::cout << "Iteration " << k << std::endl;

        const auto start = omp_get_wtime();

        multiply<<<blocks, threads>>>(r.data().get(), diag_gpu.data().get(), z.data().get(), N);
        getLastCudaError("Kernel execution failed");

        rho[0] = rho[1];

        // std::cout << r.size() << std::endl << z.size() << std::endl << N << std::endl;

        assert(r.size() == z.size() && r.size() == N && "Size mismatch");
        dot_gpu(threads, blocks, r.data().get(), z.data().get(), vec_buf.data().get(), rho.data().get() + 1, N);
        getLastCudaError("Kernel execution failed");

        // rho[1] = *buf_gpu;

        if (k == 1)
            p = z;
        else {
            const float beta = rho[1] / rho[0];
            // TODO: move to GPU (?)
            // axpy(beta, p, z, N, p);
            axpy<<<blocks, threads>>>(beta, p.data().get(), z.data().get(), N, p.data().get());
            getLastCudaError("Kernel execution failed");

        }

        spMV<<<blocks, threads>>>(ia_gpu.data().get(), ja_gpu.data().get(), a_gpu.data().get(), p.data().get(), q.data().get(), N);
        getLastCudaError("Kernel execution failed");

        assert(p.size() == q.size() && p.size() == N && "Size mismatch");
        dot_gpu(threads, blocks, p.data().get(), q.data().get(), vec_buf.data().get(), buf_gpu, N);
        getLastCudaError("Kernel execution failed");

        hipMemcpy(&buf, buf_gpu, sizeof(float), hipMemcpyDeviceToHost);
        float alpha = rho[1] / buf;

        axpy<<<blocks, threads>>>(alpha, p.data().get(), x.data().get(), N, x.data().get());
        getLastCudaError("Kernel execution failed");

        axpy<<<blocks, threads>>>(-alpha, q.data().get(), r.data().get(), N, r.data().get());
        getLastCudaError("Kernel execution failed");


        LOG_INFO << "Time " << omp_get_wtime() - start << std::endl;
        // LOG_INFO << "rho = " << rho[0] << ", " << rho[1] << ", alpha = " << alpha << std::endl;
        dot_gpu(threads, blocks, x.data().get(), x.data().get(), vec_buf.data().get(), norm_gpu, N);
        // dot<<<blocks, threads>>>(x.data().get(), x.data().get(), norm_gpu, N);
        hipMemcpy(&norm, norm_gpu, sizeof(float), hipMemcpyDeviceToHost);
        LOG_INFO << "L2 norm: " << std::sqrt(norm) << std::endl;
        LOG << "--------------------------------------------------" << std::endl << std::endl;

    }
    while (rho[1] > eps * eps && k < maxit);

    // #pragma omp parallel for default(none) shared(res, x, N)
    hipMemcpy(res, x.data().get(), N * sizeof(float), hipMemcpyDeviceToHost);
    std::cout << x[0] << x[1] << std::endl;

    // delete[] z;
    // delete[] p;
    // delete[] q;
    // delete[] x;
    // delete[] r;
    // delete[] rho;

    hipFree(buf_gpu);
    hipFree(norm_gpu);

    return k;
}
