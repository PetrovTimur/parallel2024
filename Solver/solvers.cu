#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "thrust/device_vector.h"
#include "Solver/Kernels/mathfunc.cuh"
#include <Utilities/logger.h>
#include <Utilities/cuda_helper.cuh>
#include "solvers.cuh"
#include "omp.h"


int solve(const int *ia, const int *ja, const float *a, const float *b, const float *diag, int size, float *res, const double eps, const int maxit) {
    const int N = size;
    int blocks = 204;
    int threads = 256;

    getDeviceSpecs(blocks, threads);

    LOG_INFO << "Using " << blocks << " blocks, " << threads << " threads/block" << std::endl;

    thrust::device_vector<float> z(N);
    thrust::device_vector<float> p(N);
    thrust::device_vector<float> q(N);
    thrust::device_vector<float> x(N);
    thrust::device_vector<float> r(b, b + N);
    thrust::device_vector<float> diag_gpu(diag, diag + N);

    thrust::device_vector<float> rho(2);
    thrust::device_vector<int> ia_gpu(ia, ia + N + 1);
    thrust::device_vector<int> ja_gpu(ja, ja + ia[N]);
    thrust::device_vector<float> a_gpu(a, a + ia[N]);

    thrust::device_vector<float> vec_buf(blocks);

    float *buf_gpu, *norm_gpu;
    hipMalloc(&buf_gpu, sizeof(float));
    hipMalloc(&norm_gpu, sizeof(float));
    int k = 0;
    float buf, norm;

    // std::cout << "Starting: ";

    do {
        k++;

        LOG_INFO << "Iteration " << k << std::endl;
        // std::cout << "Iteration " << k << std::endl;

        const auto start = omp_get_wtime();

        multiply<<<blocks, threads>>>(r.data().get(), diag_gpu.data().get(), z.data().get(), N);
        getLastCudaError("Kernel execution failed");

        rho[0] = rho[1];

        // std::cout << r.size() << std::endl << z.size() << std::endl << N << std::endl;

        assert(r.size() == z.size() && r.size() == N && "Size mismatch");
        dot_gpu(threads, blocks, r.data().get(), z.data().get(), vec_buf.data().get(), rho.data().get() + 1, N);
        getLastCudaError("Kernel execution failed");

        // rho[1] = *buf_gpu;

        if (k == 1)
            p = z;
        else {
            const float beta = rho[1] / rho[0];
            // TODO: move to GPU (?)
            // axpy(beta, p, z, N, p);
            axpy<<<blocks, threads>>>(beta, p.data().get(), z.data().get(), N, p.data().get());
            getLastCudaError("Kernel execution failed");

        }

        spMV<<<blocks, threads>>>(ia_gpu.data().get(), ja_gpu.data().get(), a_gpu.data().get(), p.data().get(), q.data().get(), N);
        getLastCudaError("Kernel execution failed");

        assert(p.size() == q.size() && p.size() == N && "Size mismatch");
        dot_gpu(threads, blocks, p.data().get(), q.data().get(), vec_buf.data().get(), buf_gpu, N);
        getLastCudaError("Kernel execution failed");

        hipMemcpy(&buf, buf_gpu, sizeof(float), hipMemcpyDeviceToHost);
        float alpha = rho[1] / buf;

        axpy<<<blocks, threads>>>(alpha, p.data().get(), x.data().get(), N, x.data().get());
        getLastCudaError("Kernel execution failed");

        axpy<<<blocks, threads>>>(-alpha, q.data().get(), r.data().get(), N, r.data().get());
        getLastCudaError("Kernel execution failed");


        LOG_INFO << "Time " << omp_get_wtime() - start << std::endl;
        // LOG_INFO << "rho = " << rho[0] << ", " << rho[1] << ", alpha = " << alpha << std::endl;
        dot_gpu(threads, blocks, x.data().get(), x.data().get(), vec_buf.data().get(), norm_gpu, N);
        // dot<<<blocks, threads>>>(x.data().get(), x.data().get(), norm_gpu, N);
        hipMemcpy(&norm, norm_gpu, sizeof(float), hipMemcpyDeviceToHost);
        LOG_INFO << "Solution norm: " << std::sqrt(norm) << std::endl;
        LOG << "--------------------------------------------------" << std::endl << std::endl;

    }
    while (rho[1] > eps * eps && k < maxit);

    hipMemcpy(res, x.data().get(), N * sizeof(float), hipMemcpyDeviceToHost);

    // delete[] z;
    // delete[] p;
    // delete[] q;
    // delete[] x;
    // delete[] r;
    // delete[] rho;

    hipFree(buf_gpu);
    hipFree(norm_gpu);

    return k;
}
